#define _USE_MATH_DEFINES


#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <complex>
#include <hip/hip_complex.h>
#include <assert.h>

#include <hipsparse.h>
using namespace std;

// Sparse matrix reserves the first element for dimension, assuming matrix is square.
// It is implemented using a vector.
struct sparse_elt {
    long u;
    long v;
    hipDoubleComplex amp;
};

__device__ sparse_elt
createElt(int u, int v, hipDoubleComplex amp) {
    sparse_elt elt;
    elt.u = u;
    elt.v = v;
    elt.amp = amp;
    return elt;
}

__device__ double
norm(hipDoubleComplex amp) {
    return hipCreal(hipCmul(amp, hipConj(amp)));
}

double normHost(hipDoubleComplex amp) {
    return hipCreal(hipCmul(amp, hipConj(amp)));
}

__device__ bool
equalsZero(hipDoubleComplex amplitude) {
    return (norm(amplitude) < 1e-15);
}

bool equalsZeroHost(hipDoubleComplex amplitude) {
    return (normHost(amplitude) < 1e-15);
}


__global__ void
assignAll(hipDoubleComplex *vec, long N, hipDoubleComplex val) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int numPerThread = max(N / (blockDim.x * gridDim.x), 1L);
    for(int i = id * numPerThread; i < id * (numPerThread + 1) && i < N; ++i) {
        vec[id] = val;
    }
}

__device__ void
assignAllDoublesDevice(double *vec, long N, double val) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int numPerThread = max(N / (blockDim.x * gridDim.x), 1L);
    for(int i = id * numPerThread; i < id * (numPerThread + 1) && i < N; ++i) {
        vec[id] = val;
    }
}

__global__ void
assignOne(hipDoubleComplex *vec, long N, hipDoubleComplex val, long which) {
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        vec[which] = val;
    }
}

__device__ void
sumReductionHelper(int id, double *vec, double *ans){
    atomicAdd(ans, vec[id]);
}

__global__ void
totalMag (hipDoubleComplex *state, long N, double *temp, double *ans) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int numPerThread = max(N / (blockDim.x * gridDim.x), 1L);
    assignAllDoublesDevice(temp, blockDim.x * gridDim.x, 0.0);
    double sum = 0;
    for (int i = id * numPerThread; i < id * (numPerThread + 1) && i < N; ++i) {
        sum += norm(state[i]);
    }
    temp[id] = sum;
    __syncthreads();
    sumReductionHelper(id, temp, ans);
}


bool isNormalized(hipDoubleComplex *state, const long N, int blocks, int threadsPerBlock) {
    double* temp;
    double* ansDevice;
    hipMalloc((void **)&temp, sizeof(double) * blocks * threadsPerBlock);
    hipMalloc((void **)&ansDevice, sizeof(double));
    totalMag<<<blocks, threadsPerBlock>>> (state, N, temp, ansDevice);
    double ansHost = 0.0;
    hipMemcpy(&ansHost, ansDevice, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(temp);
    hipFree(ansDevice);
    return (abs(ansHost - 1.0) < 1e-13);
}

hipDoubleComplex* uniform(const long N, int blocks, int threadsPerBlock) {
    hipDoubleComplex* state;
    hipMalloc((void **)&state, sizeof(hipDoubleComplex) * N);
    assignAll<<<blocks, threadsPerBlock>>> (state, N, make_hipDoubleComplex(sqrt(1.0/N),0.0));
    
    return state;
}
hipDoubleComplex* zero(const long N, int blocks, int threadsPerBlock) {
    hipDoubleComplex* state;
    hipMalloc((void **)&state, sizeof(hipDoubleComplex) * N);
    assignAll<<<blocks, threadsPerBlock>>>(state, N, make_hipDoubleComplex(0.0, 0.0));
    return state;
}

hipDoubleComplex* classical(const long N, int which, int blocks, int threadsPerBlock){
    hipDoubleComplex* state;
    hipMalloc((void **)&state, sizeof(hipDoubleComplex) * N);
    assignOne<<<blocks, threadsPerBlock>>>(state, N, make_hipDoubleComplex(1.0, 0.0), which);
    return state;
}
/*
cuDoublecomplex* applyOperator(vector<complex<double> > state, vector<sparse_elt> unitary) {
    //assert(unitary.begin()->u == state.size());
    vector<complex<double> > newstate = zero(state.size());
    for(auto i = unitary.begin() + 1; i != unitary.end(); ++i) {
        newstate.data()[i->u] += i->amp * state.data()[i->v];
    }
    return newstate;
}

vector<complex<double> > applyOperator(vector<complex<double> > state, vector<sparse_elt> unitary) {
    assert(unitary.begin()->u == state.size());
    vector<complex<double> > newstate = zero(state.size());
    for(auto i = unitary.begin() + 1; i != unitary.end(); ++i) {
        newstate.data()[i->u] += i->amp * state.data()[i->v];
    }
    return newstate;
}

vector<sparse_elt> tensor(vector<sparse_elt> u1, vector<sparse_elt> u2) {
    int dim1 = u1.begin()->u;
    int dim2 = u2.begin()->u;
    vector<sparse_elt> u3;
    u3.push_back(createElt(dim1*dim2, dim1*dim2, 0.0));
    for(auto i = u1.begin() + 1; i != u1.end(); ++i) {
        for(auto j = u2.begin() + 1; j != u2.end(); ++j) {
            u3.push_back(createElt(i->u * dim2 + j->u, i->v * dim2 + j->v, i->amp * j->amp));
        }
    }
    return u3;
}
*/
/* Print features */

void printComplex(hipDoubleComplex amplitude) {
    cout << "(" << hipCreal(amplitude) << "+" << hipCimag(amplitude) << "i)";
}
void printVec(hipDoubleComplex* state, const int n, const long N, bool printAll) {
    hipDoubleComplex* stateHost = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex) * N);
    hipMemcpy(stateHost, state, sizeof(hipDoubleComplex) * N, hipMemcpyDeviceToHost);
    cout << "State:\n";
    for (int i = 0; i < N; ++i) {
        if (!printAll && equalsZeroHost(stateHost[i]))
            continue;
        printComplex(stateHost[i]);
        cout << "|";
        for (int j = n-1; j >= 0; --j) {
            cout << (0x1 & (i>>j));
        }
        cout << "> + ";
        cout << "\n";
    }
    free(stateHost);
}
void printVecProbs(hipDoubleComplex* state, const int n, const long N, bool printAll) {
    hipDoubleComplex* stateHost = (hipDoubleComplex*) malloc(sizeof(hipDoubleComplex) * N);
    hipMemcpy(stateHost, state, sizeof(hipDoubleComplex) * N, hipMemcpyDeviceToHost);
    cout << "Probability distribution:\n";
    for (int i = 0; i < N; ++i) {
        if (!printAll && equalsZeroHost(stateHost[i]))
            continue;

        cout << "|<";
        for (int j = n-1; j >= 0; --j) {
            cout << (0x1 & (i>>j));
        }
        cout << "|Psi>|^2 = " << normHost(stateHost[i]) << "\n";
    }
    free(stateHost);
}
/*
void printOp(vector<sparse_elt> U) {
    vector<vector<complex<double> > > densified;
    int dim = U.begin()->u;
    densified.assign(dim, zero(dim));
    for (auto x = U.begin()+1; x != U.end(); ++x) {
        densified.data()[x->u].data()[x->v] += x->amp;
    }
    for (int x = 0; x < dim; ++x) {
        for (int y = 0; y < dim; ++y) {
            cout << densified.data()[x].data()[y] << " ";
        }
        cout << "\n";
    }
}

long getMostLikely(vector<complex<double> > state, int n) {
    long ans = -1;
    double prob = 0.0;
    long N = pow(2, n);
    for (long i = 0; i < N; i++) {
        if (norm(state.data()[i]) > prob) {
            prob = norm(state.data()[i]);
            ans = i;
        }
    }
    return ans;
}*/

/* One qubit gates */ 
/*
vector<sparse_elt> identity(){
    vector<sparse_elt> id;
    // dimensions
    id.push_back(createElt(2, 2, 0.0));
    // values
    id.push_back(createElt(0, 0, 1.0));
    id.push_back(createElt(1, 1, 1.0));
    return id;
}

vector<sparse_elt> naught() {
    vector<sparse_elt> naught;
    // dimensions
    naught.push_back(createElt(2, 2, 0.0));
    // values
    naught.push_back(createElt(1, 0, 1.0));
    naught.push_back(createElt(0, 1, 1.0));
    return naught;
}
vector<sparse_elt> hadamard() {
    vector<sparse_elt> had;
    // dimensions
    had.push_back(createElt(2, 2, 0.0));
    // values
    double norm = 1.0 / sqrt(2);
    had.push_back(createElt(0, 0, norm));
    had.push_back(createElt(0, 1, norm));
    had.push_back(createElt(1, 0, norm));
    had.push_back(createElt(1, 1, -norm));
    return had;
}
vector<sparse_elt> phase(double phi){
    vector<sparse_elt> phase;
    // dimensions
    phase.push_back(createElt(2,2, 0.0));
    // values
    phase.push_back(createElt(0,0, 1.0));
    complex<double> ex (cos(phi), sin(phi));
    phase.push_back(createElt(1,1,ex));
    return phase;
}
*/

/* Expanding operations for 1 qubit gates. */

/* It is much more efficient to do many 1 qubit gates sequentially than to compute their tensor product */
/*
vector<sparse_elt> oneQubitGateExpand(vector<sparse_elt> oneQubitGate, const int n, const int which) {
    vector<sparse_elt> total = (which == 0 ? oneQubitGate : identity());
    for(int i = 1; i < n; ++i) {
        if(which == i) 
            total = tensor(total, oneQubitGate);
        else
            total = tensor(total, identity());
    }
    return total;
}
*/

/* Two qubit gates */
/*
vector<sparse_elt> CNOTExpanded(const int n, const int u, const int v) { 
    vector<sparse_elt> cnot;
    long N = pow(2, n);
    cnot.push_back(createElt(N, N, 0.0));
    int j;
    for(int i = 0; i < N; ++i) {
        if (i>>(n-u-1) & 0x1) {
            j = i ^ (0x1<<(n-v-1));
            cnot.push_back(createElt(i, j, 1.0));
        } else {
            cnot.push_back(createElt(i, i, 1.0));
        }
    }
    return cnot;
}
vector<sparse_elt> CU1Expanded(vector<sparse_elt> U1, const int n, const int u, const int v) {
    vector<sparse_elt> CU1;
    long N = pow(2, n);
    CU1.push_back(createElt(N, N, 0.0));
    vector<complex<double> > stateOfV;
    for(long i = 0; i < N; ++i) {
        if (i>>(n-u-1) & 0x1) {
            stateOfV = classical(2, (i>>(n-v-1)) & 0x1);
            stateOfV = applyOperator(stateOfV, U1);
            if(!equalsZero(stateOfV.data()[0])) {
                CU1.push_back(createElt(i, i & ~(0x1L<<(n-v-1)), stateOfV.data()[0]));
            }
            if(!equalsZero(stateOfV.data()[1])) {
                CU1.push_back(createElt(i, i | (0x1L<<(n-v-1)), stateOfV.data()[1]));
            }
        } else {
            CU1.push_back(createElt(i, i, 1.0));
        }
    }
    return CU1;
}
vector<sparse_elt> CPhase(const int n, const int u, const int v, double phi) {
    return CU1Expanded(phase(phi), n, u, v);
}

vector<sparse_elt> swapExpanded(const int n, const int u, const int v) {
    assert(u != v);
    vector<sparse_elt> swp;
    long N = pow(2, n);
    //dimensions
    swp.push_back(createElt(N, N, 0.0));
    // values
    int j;
    for(int i = 0; i < N; ++i) {
        if ((i>>(n-u-1) & 0x1) ^ (i>>(n-v-1) & 0x1)) {
            j = (i ^ (0x1<<(n-u-1))) ^ (0x1<<(n-v-1));
            swp.push_back(createElt(i, j, 1.0));
        } else {
            swp.push_back(createElt(i, i, 1.0));
        }
    }
    return swp;
}
*/
/* Pi estimation functions for benchmarking performance. */

/* Inverse quantum fourier transform */
/*
vector<complex<double> > qft_dagger(vector<complex<double> > state, const int n) {
    const int n_prime = n+1;
    for (int i = 0; i < n/2; ++i) {
        state = applyOperator(state, swapExpanded(n_prime, i, n-i-1));    
    }
    for (int j = 0; j < n; ++j) {
        for (int m = 0; m < j; ++m) {
            double phi = -M_PI / ((double)pow(2, j - m));
            state = applyOperator(state, CPhase(n_prime, j, m, phi)); 
        }
        state = applyOperator(state, oneQubitGateExpand(hadamard(), n_prime, j));
    }
    return state;
}

// Setup for quantum phase estimation. 
cuDoubleComplex* qpe_pre(const int n){
    const long N = pow(2, n+1);
    const int n_prime = n+1;
    cuDoubleComplex* state = classical(N, 0);
    for (int i = 0; i < n; ++i) {
        state = applyOperator(state, oneQubitGateExpand(hadamard(), n_prime, i));    
    }
    state = applyOperator(state, oneQubitGateExpand(naught(), n_prime, n));

    for (int i = n-1; i >= 0; --i) {
        for (int j = 0; j < pow(2, n-i-1); ++j) {
            state = applyOperator(state, CPhase(n_prime, n, n-i-1, 1.0));
        }
    }
    return state;
}*/
/* The bits we want for this task are from 1 to n inclusive, since the 0 bit is our extra for
 * setting up the problem. Additionally, we want to read them in reverse. */
long getCorrectBitsForPiEstimate(long bits, int n){
    long answer = 0;
    for(int i = 1; i <= n; i++){
        answer = answer<<1;
        answer += (bits>>i) & 0x1L;
    }
    return answer;
}

int get_pi_estimate(const int n, const int N, int blocks, int threadsPerBlock){
    //cuDoubleComplex* state = uniform(N, blocks, threadsPerBlock);
    hipDoubleComplex* state = classical(N, 0, blocks, threadsPerBlock);
    cout << isNormalized(state, N, blocks, threadsPerBlock) << endl;
    
    //printVec(state, n, N, false);
    //printVecProbs(state, n, N, false);
    
    hipFree(state);
    return 0;
    
    /* Uniform superposition: */
    /*for (int i = 0; i < n; ++i) {
        state = applyOperator(state, oneQubitGateExpand(hadamard(), n, i));    
    }
    //state = applyOperator(state, oneQubitGateExpand(hadamard(), n, 0));
    state = applyOperator(state, oneQubitGateExpand(naught(), n, 0));
    state = applyOperator(state, CPhase(n, 0,1, 0.1));

    printVec(state, n, N, false);*/
    //printVecProbs(state, n, N, false);
}

int main(){
    int n = 3;
    long N = pow(2, n);
    int threadsPerBlock = 256;
    int blocks = (threadsPerBlock + N - 1)/threadsPerBlock;
    cout << get_pi_estimate(n, N, blocks, threadsPerBlock) << endl;

    /*cuDoubleComplex* uni = uniform(N,blocks,threadsPerBlock)
    
    for(int i = 0; i < N; i++){
        cout << uni[i] << endl;
    }*/
}
